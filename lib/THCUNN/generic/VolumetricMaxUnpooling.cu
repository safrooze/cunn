
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricMaxUnpooling.cu"
#else

void THNN_(VolumetricMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  THCUNN_assertSameGPU_generic(state, 3, input, indices, output);

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }
  else
  {
    THArgCheck(false, 2, "4D or 5D tensor expected");
  }

  if (input->nDimension == 4) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }
  else
  { /* 5D */
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  THCTensor_(zero)(state, output);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaInput;
  THCDeviceTensor<real, 4> cudaOutput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaInput  = toDeviceTensor<real, 4>(state, input);
    cudaOutput = toDeviceTensor<real, 4>(state, output);
    cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);
  }
  else
  {
    cudaInput  = toDeviceTensor<real, 5>(state, input).downcastOuter<4>();
    cudaOutput = toDeviceTensor<real, 5>(state, output).downcastOuter<4>();
    cudaIndices = toDeviceTensor<THCIndex_t, 5>(state, indices).downcastOuter<4>();
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateOutput<<<grid, block,
          0, THCState_getCurrentStream(state)>>>(
                             cudaInput, cudaIndices, cudaOutput,
                             dT, dH, dW,
                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
}

void THNN_(VolumetricMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  THCUNN_assertSameGPU_generic(state, 4, input, indices, gradOutput, gradInput);

  if (THCTensor_(nDimension)(state, input) == 4) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaGradInput;
  THCDeviceTensor<real, 4> cudaGradOutput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaGradInput  = toDeviceTensor<real, 4>(state, gradInput);
    cudaGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
    cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);
  }
  else
  {
    cudaGradInput =
      toDeviceTensor<real, 5>(state, gradInput).downcastOuter<4>();
    cudaGradOutput =
      toDeviceTensor<real, 5>(state, gradOutput).downcastOuter<4>();
    cudaIndices =
      toDeviceTensor<THCIndex_t, 5>(state, indices).downcastOuter<4>();
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateGradInput<<<grid, block,
      0, THCState_getCurrentStream(state)>>>(
                                             cudaGradOutput,
                                             cudaIndices,
                                             cudaGradInput,
                                             dT, dH, dW,
                                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  // cleanup
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCIndexTensor_(free)(state, indices);
}

#endif
